/*
 * nvidia K80
 * Total amount of shared memory per block:       49152 bytes
 * Warp Size:                     32
 * Maximum Threads per Block:     1024
 * Maximum Block Dimensions:      1024, 1024, 64
 * Maximum Grid Dimensions:       2147483647 x 65535 x 65535
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define idx(JMAX, I, J) ((JMAX)*(I)+(J))

__device__ static void clearbuf(size_t *dsize, double *p) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
    if (i >= *dsize || j >= *dsize) return;
    p[idx(*dsize, i, j)] = 0.0;
}

/* with shared memory
 * ref: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared-memory 
 */
#ifdef _USE_SM
__global__ static void _mydgemm(size_t *dsize, double *dA, double *dB, double *dC) {
    unsigned int k, ii;
    unsigned int bi, bj, ti, tj;
    unsigned int subsize, stride;
#if 0
    extern __shared__ double dAsub[], dBsub[]; // this does not work!
#endif
    /* http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared */
    extern __shared__ double sm[];
    double *dAsub = &sm[0];
    double *dBsub = &sm[blockDim.x*blockDim.y];
    double *pdAsub, *pdBsub, *pdCsub;

    bi = blockIdx.x;
    bj = blockIdx.y;
    ti = threadIdx.x;
    tj = threadIdx.y;

    if (blockDim.x != blockDim.y || gridDim.x != gridDim.y) return;
    subsize = blockDim.x;
    stride = *dsize;

    clearbuf(dsize, dC);

    pdCsub = &dC[subsize*idx(stride, bi, bj)];
    pdCsub[idx(stride, ti, tj)] = 0.0;
    for (ii=0; ii<gridDim.x; ii++) {
	pdAsub = &dA[subsize*idx(stride, bi, ii)];
	pdBsub = &dB[subsize*idx(stride, ii, bj)];
	/* copy the elements to the shared memory */
	dAsub[idx(subsize, ti, tj)] = pdAsub[idx(stride, ti, tj)];
	dBsub[idx(subsize, ti, tj)] = pdBsub[idx(stride, ti, tj)];
	__syncthreads();
	for (k=0; k<subsize; k++)
	    pdCsub[idx(stride, tj, ti)] += dAsub[idx(subsize, tj, k)]*dBsub[idx(subsize, k, ti)];
	__syncthreads();
    }
}
#else
/* no shared memory */
__global__ static void _mydgemm(size_t *dsize, double *dA, double *dB, double *dC) {
    unsigned int i, j, k;
    i = blockIdx.x*blockDim.x + threadIdx.x;
    j = blockIdx.y*blockDim.y + threadIdx.y;
    clearbuf(dsize, dC);
    if (i >= *dsize || j >= *dsize) return;
    for (k=0; k<*dsize; k++)
        dC[idx(*dsize, j, i)] += dA[idx(*dsize, j, k)]*dB[idx(*dsize, k, i)];
}
#endif /* _USE_SM */

__host__ void mydgemm(dim3 &nblocks_per_grid, dim3 &nthreads_per_block, size_t size, double *hA, double *hB, double *hC) {
    double *dA, *dB, *dC;
    size_t *dsize;
    hipDeviceProp_t dp;
    unsigned int warpsize, smsize, smsize_used;

    hipGetDeviceProperties(&dp, 0);
    warpsize = dp.warpSize;
    smsize   = dp.sharedMemPerBlock;
//    printf("warp size: %u\n", warpsize);

    printf("# of blocks per grid:   x: %u, y: %u\n", nblocks_per_grid.x,   nblocks_per_grid.y);
    printf("# of threads per block: x: %u, y: %u\n", nthreads_per_block.x, nthreads_per_block.y);
    if (nthreads_per_block.x*nthreads_per_block.y > dp.maxThreadsPerBlock)
	printf("warning: nthreads_per_block.x*nthreads_per_block.y exceeds dp.maxThreadsPerBlock, dp.maxThreadsPerBlock: %u\n", dp.maxThreadsPerBlock);
    
    hipMalloc((void**)&dA,    sizeof(*dA)*size*size);
    hipMalloc((void**)&dB,    sizeof(*dB)*size*size);
    hipMalloc((void**)&dC,    sizeof(*dC)*size*size);
    hipMalloc((void**)&dsize, sizeof(*dsize));
    
    hipMemcpy(dA,    hA,    sizeof(*dA)*size*size, hipMemcpyHostToDevice);
    hipMemcpy(dB,    hB,    sizeof(*dA)*size*size, hipMemcpyHostToDevice);
    hipMemcpy(dsize, &size, sizeof(*dsize),        hipMemcpyHostToDevice);

#ifdef _USE_SM
    smsize_used = sizeof(*dA)*nthreads_per_block.x*nthreads_per_block.y*2;
    if (smsize_used >= smsize)
	printf("warning: used shared memory exceeds the limit, used shared memory size[B]:%u limit[B]: %u\n", smsize_used, smsize);
    printf("shared memory version\nsize of shared memory used[B]: %u\n", smsize_used);
    _mydgemm<<<nblocks_per_grid, nthreads_per_block, smsize_used>>>(dsize, dA, dB, dC);
#else
    printf("no shared memory version\n");
    _mydgemm<<<nblocks_per_grid, nthreads_per_block>>>(dsize, dA, dB, dC);
#endif /* _USE_SM */

    
    hipMemcpy(hC, dC, sizeof(*hC)*size*size, hipMemcpyDeviceToHost);
    hipFree((void*)dA);
    hipFree((void*)dB);
    hipFree((void*)dC);
    hipFree((void*)dsize);
}
